#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void add_vec(float *d_a, int n)
{
	int id = blockIdx.x*blockDim.x+threadIdx.x;
	if(id<n) d_a[n-1]+=d_a[id];	
}
int main()
{
	int i,n=10;
	float h_a[n+1];
	h_a[n]=0;
	for(i=0;i<n;i++)
		scanf("%f",&h_a[i]);
	n++;
	float *d_a;
        hipMalloc((void**)&d_a,n*sizeof(float));
        hipMemcpy(d_a,h_a,n*sizeof(float),hipMemcpyHostToDevice);
        add_vec<<<2,5>>>(d_a,n);
	hipMemcpy(h_a,d_a,n*sizeof(float),hipMemcpyDeviceToHost);
	printf("The sum is : %f \n",h_a[n-1]);
	hipFree(d_a);
	return 0;
}
