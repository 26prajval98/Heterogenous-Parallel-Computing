#include<stdio.h>
#include<hip/hip_runtime.h>
__global__ void add_vec(float *d_a, int n)                           //7.CUDA Kernel that computes sum
{
	int i = threadIdx.x;
	if((n-i-1)!=i)	
	{
		d_a[i]+=d_a[n-i-1];
	}
}
int main()
{
	int i,n;
	printf("Enter the size of the array: ");
	scanf("%d",&n);
	float *h_a;
	h_a = (float *)malloc(sizeof(float)*n);
	printf("Enter the values of the array: ");
	for(i=0;i<n;i++)
		scanf("%f",&h_a[i]);
	float *d_a; 
        hipMalloc((void**)&d_a,n*sizeof(float));                    //1.Allocate device memory
        hipMemcpy(d_a,h_a,n*sizeof(float),hipMemcpyHostToDevice);  //2.Copy host memory to device
        int Block_size = 1, threads_used;                            //3.Initialise thread block and kernel grid dimensions
	for(i=n;i>1;i= i%2?i/2 + 1:i/2)
	{
		threads_used = i/2;                                 
		add_vec<<<Block_size,threads_used>>>(d_a,i);         //4.Invoke kernel
	}
	float z;
	hipMemcpy(&z,&d_a[0],sizeof(float),hipMemcpyDeviceToHost); //5.Copy results from device to host 
	hipFree(d_a);                                               //6.Free device memory
	free(h_a);                         			     // free host memory
	printf("The sum is : %f \n",z);                                               
	return 0;
}
