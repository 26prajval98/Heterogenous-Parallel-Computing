#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
__global__ void Matrix_Add(float *d_m, float *d_n, float *d_s, long long int a, long long int b)
{
	long long int i = blockIdx.y*blockDim.y + threadIdx.y;
	long long int j = blockIdx.x*blockDim.x + threadIdx.x;
	
	if(i<a)
	{
		if(j<b)
			*(d_s + i*a + j) = *(d_m + i*a +j) + *(d_n + i*a + j);
	}
	
}
int main()
{
	long long int a,b,i,j;
	printf("Enter the dimensions of the 2 matrices: ");
	scanf("%lld %lld",&a,&b);                             //Matrices are aXb
	float *h_m = (float *)malloc(a*b*sizeof(float));
	float *h_n = (float *)malloc(b*a*sizeof(float));
	float *h_s = (float *)malloc(a*b*sizeof(float));
	srand((unsigned int)time(NULL));                      //Seeding the random function
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			*(h_m + i*a +j) = ((float)rand());
	}
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			*(h_n + i*b +j) = ((float)rand());
	}
	
	dim3 DimBlock(16,16,1);
	dim3 DimGrid(ceil(n/16),ceil(n/16),1);

	hipMalloc((void**)&d_m,a*b*sizeof(float));
	hipMalloc((void**)&d_n,b*a*sizeof(float));
	hipMalloc((void**)&d_s,a*b*sizeof(float));

	hipMemcpy(d_m,h_m,a*b*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_n,h_n,b*a*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_s,h_s,a*b*sizeof(float),hipMemcpyHostToDevice);
	 
	Matrix_Add<<<DimGrid,DimBlock>>(*d_m,*d_n,*d_s,a,b);
	
	hipMemcpy(h_m,d_m,a*b*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_n,d_n,b*a*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_s,d_s,a*b*sizeof(float),hipMemcpyDeviceToHost);
	
	hipFree(d_m); hipFree(d_n); hipFree(d_s); 
	
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			printf("%f ",*(h_s + i*a + j));
		printf("\n");
	}
	
	free(h_m); free(h_n); free(h_s);
	return 0;
}
