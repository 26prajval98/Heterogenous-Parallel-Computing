#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

int main()
{
	long long int a,b,c,i,j;
	printf("Enter the dimensions of the 2 matrices: ");
	scanf("%lld %lld %lld",&a,&b,&c);                     //Matrices are aXb and bXc
	float *h_m = (float *)malloc(a*b*sizeof(float));
	float *h_n = (float *)malloc(b*c*sizeof(float));
	float *h_s = (float *)malloc(a*c*sizeof(float));      //aXb * bXc = aXc
	srand((unsigned int)time(NULL));                      //Seeding the random function
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			*(h_m + i*a +j) = ((float)rand());
	}
	for(i=0;i<b;i++)
	{
		for(j=0;j<c;j++)
			*(h_n + i*b +j) = ((float)rand());
	}
	dim3 DimBlock(16,16,1);
	dim3 DimGrid(ceil(n/16),ceil(n/16),1);

	hipMalloc((void**)&d_m,a*b*sizeof(float));
	hipMalloc((void**)&d_n,b*c*sizeof(float));
	cudaMallic((void**)&d_s,a*c*sizeof(float));

	hipMemcpy(d_m,h_m,a*b*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_n,h_n,b*c*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_s,h_s,a*c*sizeof(float),hipMemcpyHostToDevice);
	 
	Matrix_Mul<<<DimGrid,DimBlock>>(*d_m,*d_n,*d_s,a,b,c);
	
	hipMemcpy(h_m,d_m,a*b*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_n,d_n,b*c*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_s,d_s,a*c*sizeof(float),hipMemcpyDeviceToHost);
	
	hipFree(d_m); hipFree(d_n); hipFree(d_s); 
	
	for(i=0;i<a;i++)	
	{
		for(j=0;j<c;j++)
			printf("%f ",*(h_s + i*a + j));
		printf("\n");
	}
	
	free(h_m); free(h_n); free(h_s);
	return 0;
}
