#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
__global__ void Matrix_Add(float *d_m, float *d_n, float *d_s, long long int a, long long int b)
{
	long long int i = blockIdx.y*blockDim.y + threadIdx.y;
	long long int j = blockIdx.x*blockDim.x + threadIdx.x;
	
	if((i<a) && (j<b))
	{
		*(d_s + i*b + j) = *(d_m + i*b +j) + *(d_n + i*b + j);
	}
	
}
int main()
{
	long long int a,b,i,j;
	printf("Enter the dimensions of the 2 matrices: ");
	scanf("%lld %lld",&a,&b);                             //Matrices are aXb
	float *h_m = (float *)malloc(a*b*sizeof(float));
	float *h_n = (float *)malloc(b*a*sizeof(float));
	float *h_s = (float *)malloc(a*b*sizeof(float));
	srand((unsigned int)time(NULL));                      //Seeding the random function
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			*(h_m + i*b +j) = ((float)rand());
	}
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			*(h_n + i*b +j) = ((float)rand());
	}
	
	dim3 DimGrid(ceil(a/16),ceil(b/16),1);
	dim3 DimBlock(16,16,1);
	
	float *d_m, *d_n, *d_s;
	hipMalloc((float**)&d_m,a*b*sizeof(float));
	hipMalloc((float**)&d_n,b*a*sizeof(float));
	hipMalloc((float**)&d_s,a*b*sizeof(float));

	hipMemcpy(d_m,h_m,a*b*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_n,h_n,b*a*sizeof(float),hipMemcpyHostToDevice);
	hipMemcpy(d_s,h_s,a*b*sizeof(float),hipMemcpyHostToDevice);
	 
	Matrix_Add<<<DimGrid,DimBlock>>>(d_m,d_n,d_s,a,b);
	
	hipMemcpy(h_m,d_m,a*b*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_n,d_n,b*a*sizeof(float),hipMemcpyDeviceToHost);
        hipMemcpy(h_s,d_s,a*b*sizeof(float),hipMemcpyDeviceToHost);
	
	for(i=0;i<a;i++)
        {
                for(j=0;j<b;j++)
                        printf("%f ",*(h_s + i*b + j));
                printf("\n");
        }printf("\n");
        for(i=0;i<a;i++)
        {
                for(j=0;j<b;j++)
                        printf("%f ",*(h_m + i*b + j));
                printf("\n");
        }printf("\n");
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			printf("%f ",*(h_n + i*b + j));
		printf("\n");
	}

	hipFree(d_m); hipFree(d_n); hipFree(d_s);
	
	free(h_m); free(h_n); free(h_s);
	printf("1. How many floating operations are being performed in the matrix addition kernel? \n Ans: The number of floating point operations is %lld\n",a*b);
	printf("2. How many global memory reads are being performed by your kernel?\n Ans: The number of global memory reads is 2 per thread operation = %lld\n",2*a*b);
	printf("3. How many global memory writes are being performed by your kernel?\n Ans: The number of global memory writes is oen per thread operation = %lld\n",a*b);
	return 0;
}
