#include "hip/hip_runtime.h"
// 16CO145 Sumukha PK
// 16CO234 Prajval M

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
__global__ void Matrix_Mul(long long int *d_m, long long int *d_n, long long int *d_p, long long int a, long long int b,long long int c)
{
	long long int i = blockIdx.y*blockDim.y + threadIdx.y;
	long long int j = blockIdx.x*blockDim.x + threadIdx.x;
	long long int temp = 0;

	for(long long int k=0; k<b; k++)
		temp += d_m[i*b + k] * d_n[b*k + j]; 

	d_p[i*b + j]  =  temp;

}
int main()
{
	long long int a,b,c,i,j;
	printf("Enter the dimensions of the matrices: ");
	scanf("%lld %lld %lld",&a,&b,&c);                                                //Matrices are aXb and bXc
	long long int *h_m = (long long int *)malloc(a*b*sizeof(long long int));
	long long int *h_n = (long long int *)malloc(b*c*sizeof(long long int));
	long long int *h_p = (long long int *)malloc(a*c*sizeof(long long int));
	srand((unsigned int)time(NULL));                                                 //Seeding the random function
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			*(h_m + i*b +j) = ((long long int)rand());
	}
	for(i=0;i<b;i++)
	{
		for(j=0;j<c;j++)
			*(h_n + i*c +j) = ((long long int)rand());
	}
	
	// dim3 DimGrid(ceil(a),ceil(c),1);
	dim3 DimBlock(a,c,1);
	
	long long int *d_m, *d_n, *d_p;
	hipMalloc((long long int**)&d_m,a*b*sizeof(long long int));
	hipMalloc((long long int**)&d_n,b*c*sizeof(long long int));
	hipMalloc((long long int**)&d_p,a*c*sizeof(long long int));

	hipMemcpy(d_m,h_m,a*b*sizeof(long long int),hipMemcpyHostToDevice);
	hipMemcpy(d_n,h_n,b*c*sizeof(long long int),hipMemcpyHostToDevice);
	hipMemcpy(d_p,h_p,a*c*sizeof(long long int),hipMemcpyHostToDevice);
	 
	Matrix_Mul<<<1,DimBlock>>>(d_m,d_n,d_p,a,b,c);
	
	hipMemcpy(h_m,d_m,a*b*sizeof(long long int),hipMemcpyDeviceToHost);
    hipMemcpy(h_n,d_n,b*c*sizeof(long long int),hipMemcpyDeviceToHost);
    hipMemcpy(h_p,d_p,a*c*sizeof(long long int),hipMemcpyDeviceToHost);
	
	for(i=0;i<a;i++)
    {
        for(j=0;j<c;j++)
            printf("%lld ",*(h_p + i*c + j));
        printf("\n");
    }
    printf("\n");
    for(i=0;i<a;i++)
    {
        for(j=0;j<b;j++)
            printf("%lld ",*(h_m + i*b + j));
        printf("\n");
    }
    printf("\n");
	for(i=0;i<b;i++)
	{
		for(j=0;j<c;j++)
			printf("%lld ",*(h_n + i*c + j));
		printf("\n");
	}

	hipFree(d_m); hipFree(d_n); hipFree(d_p);
	
	free(h_m); free(h_n); free(h_p);
	return 0;
}
