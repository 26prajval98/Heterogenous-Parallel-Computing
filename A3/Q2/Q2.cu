#include "hip/hip_runtime.h"
// 16CO145 Sumukha PK
// 16CO234 Prajval M

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>

# define SIZE 16 
 
__global__ void Matrix_Mul(long long int *d_m, long long int *d_n, long long int *d_p, long long int a, long long int b,long long int c)
{
	long long int oi = blockIdx.y*blockDim.y + threadIdx.y;
	long long int oj = blockIdx.x*blockDim.x + threadIdx.x;
	long long int temp = 0;

	long long int i = min(oi, a-1);
	long long int j = min(oj, c-1);

	for(long long int k=0; k<b; k++){
		temp += d_m[i*b + k] * d_n[b*k + j]; 
	}

	d_p[i*c + j]  =  temp;
	__syncthreads();
	
}
int main()
{
	long long int a,b,c,i,j;
	printf("Enter the dimensions of the matrices: ");
	scanf("%lld %lld %lld",&a,&b,&c);                                                //Matrices are aXb and bXc
	long long int *h_m = (long long int *)malloc(a*b*sizeof(long long int));
	long long int *h_n = (long long int *)malloc(b*c*sizeof(long long int));
	long long int *h_p = (long long int *)malloc(a*c*sizeof(long long int));

	srand((unsigned int)time(NULL));                                                 //Seeding the random function

	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
			h_m[i*b + j] = ((long long int)rand());
	}

	for(i=0;i<b;i++)
	{
		for(j=0;j<c;j++)
			h_n[i*c + j] = ((long long int)rand());
	}
	
	dim3 DimGrid((ceil(a)/SIZE),(ceil(c)/SIZE),1);
	dim3 DimBlock(SIZE,SIZE,1);
	
	long long int *d_m, *d_n, *d_p;
	hipMalloc((long long int**)&d_m,a*b*sizeof(long long int));
	hipMalloc((long long int**)&d_n,b*c*sizeof(long long int));
	hipMalloc((long long int**)&d_p,a*c*sizeof(long long int));

	hipMemcpy(d_m,h_m,a*b*sizeof(long long int),hipMemcpyHostToDevice);
	hipMemcpy(d_n,h_n,b*c*sizeof(long long int),hipMemcpyHostToDevice);
	hipMemcpy(d_p,h_p,a*c*sizeof(long long int),hipMemcpyHostToDevice);
	 
	Matrix_Mul<<<1,DimBlock>>>(d_m,d_n,d_p,a,b,c);
		
	hipMemcpy(h_m,d_m,a*b*sizeof(long long int),hipMemcpyDeviceToHost);
    hipMemcpy(h_n,d_n,b*c*sizeof(long long int),hipMemcpyDeviceToHost);
    hipMemcpy(h_p,d_p,a*c*sizeof(long long int),hipMemcpyDeviceToHost);
	
	for(i=0;i<a;i++)
    {
        for(j=0;j<c;j++){
			printf("%lld ", h_p[i*c + j]);
		}
        printf("\n");
    }
    printf("\n");
    for(i=0;i<a;i++)
    {
        for(j=0;j<b;j++)
            printf("%lld ", h_m[i*b + j] );
        printf("\n");
    }
	printf("\n");
	
	for(i=0;i<b;i++)
	{
		for(j=0;j<c;j++)
			printf("%lld ", h_n[i*c + j]);
		printf("\n");
	}

	hipFree(d_m); hipFree(d_n); hipFree(d_p);
	
	free(h_m); free(h_n); free(h_p);
	return 0;
}
