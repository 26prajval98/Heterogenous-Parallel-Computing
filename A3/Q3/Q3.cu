#include "hip/hip_runtime.h"
// 16CO145 Sumukha PK
// 16CO234 Prajval M

#include<stdio.h>
#include<hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
using namespace std;

#define SIZE 32


__global__ void Matrix_Mul(long long int *d_m, long long int *d_n, long long int *d_p, long long int a, long long int b, long long int c)
{
	if(a<SIZE || c<SIZE )   													//If tiling is not necessary use older method 
	{
		long long int oj = blockIdx.y*blockDim.y + threadIdx.y;
		long long int oi = blockIdx.x*blockDim.x + threadIdx.x;
		long long int temp = 0;

		long long int i = min(oi, a-1);
		long long int j = min(oj, c-1);

		for(long long int k=0; k<b; k++)
		{
			temp += d_m[i*b + k] * d_n[k*c + j]; 
		}

		__syncthreads();

		d_p[i*c + j]  =  temp;
	}
	else
	{
		
		long long int bx = blockIdx.x;
		long long int by = blockIdx.y;
		long long int tx = threadIdx.x;
		long long int ty = threadIdx.y;
		long long int Row = by * blockDim.y + ty;
		long long int Col = bx * blockDim.x + tx;
		long long int temp = 0;

		__shared__ long long int ds_A[SIZE][SIZE];
		__shared__ long long int ds_B[SIZE][SIZE];

		long long int i = min(Row, a - 1);
		long long int j = min(Col, c - 1);

		for (long long int p = 0; p < b/SIZE; p++)
		{
		ds_A[ty][tx] = d_m[i * b + p * SIZE + tx];
		ds_B[ty][tx] = d_n[(p * SIZE + ty) * c + j];
		__syncthreads();

		for (long long int k = 0; k < SIZE; k++)
		temp += ds_A[ty][k] * ds_B[k][tx];
		__syncthreads();

		d_p[i * c + j] = temp;
		}
	}
}
void MatMul(long long int *m,long long int *n,long long int *p, long long int a,long long int b,long long int c)
{
	long long int i,j,k,f=0,sum;
	for(i=0;i<a;i++)
	{
		for(j=0;j<b;j++)
		{
			sum=0;
			for(k=0;k<c;k++)
				sum+=m[i*b+k]*n[k*c+j];
			if(sum!=p[i*b+j])
				{
					f=1;
					break;
				}
		}
	}
	if(f==1)
		printf("Error");
	else
		printf("WORKED!");
}
int main()
{
	long long int a, b, c, i, j;
	printf("Enter the dimensions of the matrices: ");
	scanf("%lld %lld %lld", &a, &b, &c); //Matrices are aXb and bXc
	long long int *h_m = (long long int *)malloc(a * b * sizeof(long long int));
	long long int *h_n = (long long int *)malloc(b * c * sizeof(long long int));
	long long int *h_p = (long long int *)malloc(a * c * sizeof(long long int));

	srand((unsigned int)time(NULL)); //Seeding the random function

	for (i = 0; i < a; i++)
	{
		for (j = 0; j < b; j++)
			h_m[i * b + j] = ((long long int)rand());
	}

	for (i = 0; i < b; i++)
	{
		for (j = 0; j < c; j++)
			h_n[i * c + j] = ((long long int)rand());
	}
	long long int c_a = a > SIZE ? (long long int)ceil(a / (float)SIZE) : 1;
	long long int c_c = c > SIZE ? (long long int)ceil(c / (float)SIZE) : 1;

	dim3 DimGrid(c_a, c_c, 1);
	dim3 DimBlock(SIZE, SIZE, 1);

	long long int *d_m, *d_n, *d_p;
	hipMalloc((long long int **)&d_m, a * b * sizeof(long long int));
	hipMalloc((long long int **)&d_n, b * c * sizeof(long long int));
	hipMalloc((long long int **)&d_p, a * c * sizeof(long long int));

	hipMemcpy(d_m, h_m, a * b * sizeof(long long int), hipMemcpyHostToDevice);
	hipMemcpy(d_n, h_n, b * c * sizeof(long long int), hipMemcpyHostToDevice);
	hipMemcpy(d_p, h_p, a * c * sizeof(long long int), hipMemcpyHostToDevice);

	Matrix_Mul<<<DimGrid, DimBlock>>>(d_m, d_n, d_p, a, b, c);

	hipMemcpy(h_m, d_m, a * b * sizeof(long long int), hipMemcpyDeviceToHost);
	hipMemcpy(h_n, d_n, b * c * sizeof(long long int), hipMemcpyDeviceToHost);
	hipMemcpy(h_p, d_p, a * c * sizeof(long long int), hipMemcpyDeviceToHost);

	for (i = 0; i < a; i++)
	{
		for (j = 0; j < c; j++)
		{
			printf("%lld ", h_p[i * c + j]);
		}
		printf("\n");
	}
	printf("\n");
	for (i = 0; i < a; i++)
	{
		for (j = 0; j < b; j++)
			printf("%lld ", h_m[i * b + j]);
		printf("\n");
	}
	printf("\n");

	for (i = 0; i < b; i++)
	{
		for (j = 0; j < c; j++)
			printf("%lld ", h_n[i * c + j]);
		printf("\n");
	}
	MatMul(h_m,h_n,h_p,a,b,c);
	hipFree(d_m);
	hipFree(d_n);
	hipFree(d_p);

	free(h_m);
	free(h_n);
	free(h_p);
	return 0;
}
