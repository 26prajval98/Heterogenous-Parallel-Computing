#include "hip/hip_runtime.h"
#include "wb.h"
#include <iostream>
#include <stdio.h>

#define NUM_BINS 4096
#define SIZE 1024
#define BIN_CAP 127

#define CUDA_CHECK(ans)                       \
	{                                         \
		gpuAssert((ans), __FILE__, __LINE__); \
	}
inline void gpuAssert(hipError_t code, const char *file, int line,
					  bool abort = true)
{
	if (code != hipSuccess)
	{
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code),
				file, line);
		if (abort)
			exit(code);
	}
}

__global__ void hist(int *d_ip, int *d_bin, int inputLength)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;

	if(idx < inputLength)
		atomicAdd(&d_bin[d_ip[idx]], 1);
}

__global__ void saturate(int * d_bin)
{
	int idx = blockIdx.x*blockDim.x + threadIdx.x;
	if(d_bin[idx] > BIN_CAP)
		d_bin[idx] = BIN_CAP;
}

int main(int argc, char *argv[])
{

	int inputLength;
	int *hostInput;
	int *hostBins;
	int *deviceInput;
	int *deviceBins;

	/* Read input arguments here */
	wbArg_t args = {argc, argv};
	wbTime_start(Generic, "Importing data and creating memory on host");
	hostInput = (int *)wbImport(wbArg_getInputFile(args, 3), &inputLength);
	hostBins = (int *)malloc(NUM_BINS * sizeof(int));

	for (int i = 0; i < NUM_BINS; i++)
		hostBins[i] = 0;

	wbTime_stop(Generic, "Importing data and creating memory on host");

	wbLog(TRACE, "The input length is ", inputLength);
	wbLog(TRACE, "The number of bins is ", NUM_BINS);

	wbTime_start(GPU, "Allocating GPU memory.");
	//@@ Allocate GPU memory here

	hipMalloc((void **)&deviceInput, inputLength * sizeof(int));
	hipMalloc((void **)&deviceBins, NUM_BINS * sizeof(int));

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Allocating GPU memory.");

	wbTime_start(GPU, "Copying input memory to the GPU.");
	//@@ Copy memory to the GPU here

	hipMemcpy(deviceInput, hostInput, inputLength * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(deviceBins, hostBins, NUM_BINS * sizeof(int), hipMemcpyHostToDevice);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(GPU, "Copying input memory to the GPU.");

	// Launch kernel
	// ----------------------------------------------------------

	wbLog(TRACE, "Launching kernel");
	wbTime_start(Compute, "Performing CUDA computation");
	//@@ Perform kernel computation here
	
	long long int d_x = inputLength > SIZE ? (long long int)ceil(inputLength/(float)SIZE) : 1;

	hist<<<d_x, SIZE>>>(deviceInput, deviceBins, inputLength);
	saturate<<<4, SIZE>>>(deviceBins);

	wbTime_stop(Compute, "Performing CUDA computation");

	wbTime_start(Copy, "Copying output memory to the CPU");
	//@@ Copy the GPU memory back to the CPU here

	hipMemcpy(hostInput, deviceInput, inputLength * sizeof(int), hipMemcpyDeviceToHost);
	hipMemcpy(hostBins, deviceBins, NUM_BINS * sizeof(int), hipMemcpyDeviceToHost);

	CUDA_CHECK(hipDeviceSynchronize());
	wbTime_stop(Copy, "Copying output memory to the CPU");

	wbTime_start(GPU, "Freeing GPU Memory");
	//@@ Free the GPU memory here

	hipFree(deviceInput);
	hipFree(deviceBins);

	wbTime_stop(GPU, "Freeing GPU Memory");

	// Verify correctness
	// -----------------------------------------------------
	wbSolution(args, hostBins, NUM_BINS);

	free(hostBins);
	free(hostInput);
	return 0;
}